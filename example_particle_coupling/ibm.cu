#include "hip/hip_runtime.h"
/*Raul P. Pelaez 2022. Particle-grid coupling example
 */
#include<uammd.cuh>
#include <misc/IBM.cuh>
#include"utils.cuh"

using namespace uammd;

// A simple Gaussian kernel compatible with the IBM module.
//A lot of them are already defined, i.e IBM_kernels::Peskin::threePoint
class Gaussian{
  const real prefactor;
  const real tau;
public:
  const int support;
  Gaussian(real width, int support):
    prefactor(pow(2.0*M_PI*width*width, -0.5)),
    tau(-0.5/(width*width)),
    support(support){}

  __device__ int3 getSupport(real3 pos, int3 cell){
    return {support, support, support};
  }

  __device__ real phi(real r, real3 pos) const{
    return prefactor*exp(tau*r*r);
  }
};


int main(){
  real L = 32;
  Box box({L,L,L});
  int3 cellDim {32,32,32};
  Grid grid(box, cellDim);

  int numberParticles = 1e6;
  int ncells = grid.getNumberCells();
  
  thrust::device_vector<real3> positions = generateRandomPositions(L, numberParticles);
  thrust::device_vector<real> particleQuantity(numberParticles), gridQuantity(ncells);
  thrust::fill(particleQuantity.begin(), particleQuantity.end(), 1);
  thrust::fill(gridQuantity.begin(), gridQuantity.end(), 0);

  const real width = 1; //An arbitrary width
  const int support = 8;//An arbitrary support
  auto kernel = std::make_shared<Gaussian>(width, support);

  IBM<Gaussian> ibm(kernel, grid);
  ibm.spread(thrust::raw_pointer_cast(positions.data()),
	     thrust::raw_pointer_cast(particleQuantity.data()),
	     thrust::raw_pointer_cast(gridQuantity.data()),
	     numberParticles);

  thrust::fill(particleQuantity.begin(), particleQuantity.end(), 0);

  ibm.gather(thrust::raw_pointer_cast(positions.data()),
	     thrust::raw_pointer_cast(particleQuantity.data()),
	     thrust::raw_pointer_cast(gridQuantity.data()),
	     numberParticles);
  //Now particleQuantity is filled with ones again
  return 0;
}
