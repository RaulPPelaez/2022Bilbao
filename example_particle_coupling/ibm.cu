#include "hip/hip_runtime.h"
/*Raul P. Pelaez 2022. Particle-grid coupling example
 */
#include<uammd.cuh>
#include <misc/IBM.cuh>
#include<thrust/random.h>
using namespace uammd;

// A simple Gaussian kernel compatible with the IBM module.
//A lot of them are already defined, i.e IBM_kernels::Peskin::threePoint
class Gaussian{
  const real prefactor;
  const real tau;
public:
  const int support;
  Gaussian(real width, int support):
    prefactor(pow(2.0*M_PI*width*width, -0.5)),
    tau(-0.5/(width*width)),
    support(support){}

  __device__ int3 getSupport(real3 pos, int3 cell){
    return {support, support, support};
  }

  __device__ real phi(real r, real3 pos) const{
    return prefactor*exp(tau*r*r);
  }
};

template<class Iter1, class Iter2>
void spreadWithIBM(Grid grid,
		   Iter1 positions,
                   Iter2 dataAtCellPositions,
                   Iter2 dataAtParticlePositions,
                   int numberParticles){
  const real width = 1; //An arbitrary width
  const int support = 8;//An arbitrary support
  auto kernel = std::make_shared<Gaussian>(width, support);
  IBM<Gaussian> ibm(kernel, grid);
  //Spreads dataAtParticlePositions into dataAtCellPositions
  ibm.spread(positions, dataAtParticlePositions, dataAtCellPositions, numberParticles);
}

template<class Iter1, class Iter2>
void interpolateWithIBM(Grid grid, Iter1 positions,
                        Iter2 dataAtCellPositions,
                        Iter2 dataAtParticlePositions,
                        int numberParticles){
  const real width = 1; //An arbitrary width
  const int support = 8;//An arbitrary support
  auto kernel = std::make_shared<Gaussian>(width, support);
  IBM<Gaussian> ibm(kernel, grid);
  //Interpolates dataAtCellPositions into dataAtParticlePositions
  ibm.gather(positions, dataAtParticlePositions, dataAtCellPositions, numberParticles);
}

//Creates and returns a vector with random positions inside a cubic box of side L (always the same random positions) 
thrust::device_vector<real3> generateRandomPositions(real L, int numberParticles){
  thrust::device_vector<real3> positions(numberParticles);
  auto it = thrust::make_counting_iterator<int>(0);
  thrust::transform(it, it+numberParticles,
		    positions.begin(),
		    [=]__device__(int i){
		      thrust::default_random_engine rng;
		      thrust::uniform_real_distribution<real> dist(-L*0.5, L*0.5);
		      rng.discard(i);
		      return make_real3(dist(rng), dist(rng), dist(rng));
		    }
		    );
  return positions;
}

int main(){
  real L = 32;
  Box box({L,L,L});
  int3 cellDim {32,32,32};
  Grid grid(box, cellDim);

  int numberParticles = 1e6;
  thrust::device_vector<real3> positions;
  positions = generateRandomPositions(L, numberParticles);
  
  
  int ncells = grid.getNumberCells();
  thrust::device_vector<real> particleQuantity(ncells), gridQuantity(ncells);
  thrust::fill(particleQuantity.begin(), particleQuantity.end(), 1);
  thrust::fill(gridQuantity.begin(), gridQuantity.end(), 0);
  
  spreadWithIBM(grid,
		thrust::raw_pointer_cast(positions.data()),
		thrust::raw_pointer_cast(gridQuantity.data()),
		thrust::raw_pointer_cast(particleQuantity.data()),
		numberParticles);
  
  thrust::fill(particleQuantity.begin(), particleQuantity.end(), 0);  
  interpolateWithIBM(grid,
		thrust::raw_pointer_cast(positions.data()),
		thrust::raw_pointer_cast(gridQuantity.data()),
		thrust::raw_pointer_cast(particleQuantity.data()),
		numberParticles);
  //Now particleQuantity is filled with ones again
  return 0;
}
