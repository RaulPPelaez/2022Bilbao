#include "hip/hip_runtime.h"
/*Raul P. Pelaez 2022. The UAMMD letters made of particles suspended in a fluid.
Runs a Brownian Hydrodynamics simulation with particles starting in a periodic box at low temperature.

You can visualize the reuslts with superpunto
*/

//This include contains the basic needs for an uammd project
#include"uammd.cuh"
#include"Integrator/BDHI/BDHI_FCM.cuh"
#include<fstream>
#include <memory>

using namespace uammd;

struct GravityAndWall : public Interactor {
  real zwall;

  GravityAndWall(std::shared_ptr<ParticleData> pd, real zwall):
    Interactor(pd, "GravityAndWall"), zwall(zwall){}
  
  void sum(Interactor::Computables comp, hipStream_t st) override{
    // bool shouldComputeForces = comp.force;
    // bool shouldComputeEnergies = comp.energy;
    // bool shouldComputeVirials = comp.virial;    
    auto pos = pd->getPos(access::gpu, access::read);
    auto force = pd->getForce(access::gpu, access::readwrite);
    real gravity = 0.1;
    real wallStrength = 1.0;
    real h = this->zwall;
    thrust::for_each_n(thrust::cuda::par.on(st),
		       thrust::make_counting_iterator<int>(0), pos.size(),
     		       [=]__device__(int i){			 
			real fz = 0;
			real pi_z = pos[i].z;
			if(pi_z<=h){
			   real distanceToWall = fabs(pi_z-h);
			   fz += wallStrength*distanceToWall*distanceToWall;
			}
			force[i].z += fz-gravity;
     		       });
  }
    
};

auto readParticles(){
  //Read contents of the file into a vector
  std::ifstream in ("pos.init");
  std::istream_iterator<real4> begin(in), end;
  std::vector<real4> h_pos(begin, end);
  int numberParticles = h_pos.size();
  //Create an UAMMD particle container
  auto pd = std::make_shared<ParticleData>(numberParticles);
  //Copy the file contents into the UAMMD positions (in the CPU)
  {
    auto pos = pd->getPos(access::cpu, access::write);
    std::copy(h_pos.begin(), h_pos.end(), pos.begin());
  }
  //Increase spatial coherence in memory
  pd->sortParticles();
  return pd;
}

auto initializeSimulation(std::shared_ptr<ParticleData> pd, Box box){
  using Scheme = BDHI::FCMIntegrator;
  Scheme::Parameters par;
  par.temperature = 1;
  par.viscosity = 1;
  par.hydrodynamicRadius =  4;
  par.dt = 0.01;
  par.box = box;
  auto integrator = std::make_shared<Scheme>(pd, par);
  return integrator;
}

void runSimulation(std::shared_ptr<ParticleData> pd, std::shared_ptr<Integrator> bdhi){
  std::ofstream out("/dev/stdout");
  Timer tim;
  tim.tic();
  int numberSteps = 2000;
  int printSteps  = 200;
  forj(0, numberSteps){
    if(j%printSteps==0){
      auto pos = pd->getPos(access::cpu, access::read);
      out<<"#"<<std::endl;
      for(auto p: pos)out<<make_real3(p)<<" 0.7 0\n";
    }
    bdhi->forwardTime();
  }
  auto totalTime = tim.toc();
  System::log<System::MESSAGE>("mean FPS: %.2f", numberSteps/totalTime);
}

int main(int argc, char *argv[]){
  {
    Box box({256, 128, 160});
    auto pd = readParticles();
    auto bdhi = initializeSimulation(pd, box);
    auto gravity = std::make_shared<GravityAndWall>(pd, -box.boxSize.z*0.5);    
    bdhi->addInteractor(gravity);
    runSimulation(pd, bdhi);
  }
  return 0;
}


// bdhi->addInteractor(createTPPoissonInteractor(pd));
//#include <Interactor/SpectralEwaldPoisson.cuh>
//  auto createTPPoissonInteractor(std::shared_ptr<ParticleData> pd){
//    {
//      auto charges = pd->getCharge(access::cpu, access::write);
//      std::fill(charges.begin(), charges.end(), 1);
//    }
//    Poisson::Parameters par;
//    par.box = Box({256, 128, 160});
//   par.epsilon = 1; //Permittivity
//   par.gw = 4.0; //Gaussian width of the sources
//   par.tolerance = 1e-2;//Overall tolerance of the algorithm
//   //If a splitting parameter is passed
//   // the code will run in Ewald split mode
//   //Otherwise, the non Ewald version will be used
//   //par.split = 1.0;
//   return std::make_shared<Poisson>(pd, par);
// }
