#include "hip/hip_runtime.h"
/*Raul P. Pelaez 2022. The UAMMD letters made of particles suspended in a fluid.
Runs a Brownian Hydrodynamics simulation with particles starting in a periodic box at low temperature.

You can visualize the reuslts with superpunto
*/

//This include contains the basic needs for an uammd project
#include "uammd.cuh"
#include "Integrator/BDHI/BDHI_EulerMaruyama.cuh"
#include "Integrator/BDHI/BDHI_FCM.cuh"
#include"Integrator/BrownianDynamics.cuh"
#include<fstream>
#include <memory>

using namespace uammd;

// A simple Interactor that sums a gravitational force to each particle.
// It also adds a potential wall at the bottom of the domain.
struct GravityAndWall : public Interactor {
  real zwall;

  GravityAndWall(std::shared_ptr<ParticleData> pd, real zwall):
    Interactor(pd, "GravityAndWall"), zwall(zwall){}
  
  void sum(Interactor::Computables comp, hipStream_t st) override{
    //This function can be called with different responsabilities.
    //Let us ignore that and compute only forces.
    // bool shouldComputeForces = comp.force;
    // bool shouldComputeEnergies = comp.energy;
    // bool shouldComputeVirials = comp.virial;
    auto pos = pd->getPos(access::gpu, access::read);
    auto force = pd->getForce(access::gpu, access::readwrite);
    real gravity = 0.1;
    real wallStrength = 1.0;
    real h = this->zwall;
    //This thrust call runs a for loop in the GPU. Iterates over all particles
    thrust::for_each_n(thrust::cuda::par.on(st),
		       thrust::make_counting_iterator<int>(0), pos.size(),
     		       [=]__device__(int i){
			real fz = 0;
			real pi_z = pos[i].z;
			//Wall repulsion
			if(pi_z<=h){
			   real distanceToWall = fabs(pi_z-h);
			   fz += wallStrength*distanceToWall*distanceToWall;
			}
			force[i].z += fz-gravity;
     		       });
  }

};

// Reads a file with positions into an UAMMD particle container and returns it
auto createParticles(){
  //Read contents of the file into a vector
  std::ifstream in ("pos.init");
  std::istream_iterator<real4> begin(in), end;
  std::vector<real4> h_pos(begin, end);
  int numberParticles = h_pos.size();
  //Create an UAMMD particle container
  auto pd = std::make_shared<ParticleData>(numberParticles);
  //Copy the file contents into the UAMMD positions (in the CPU)
  {
    auto pos = pd->getPos(access::cpu, access::write);
    std::copy(h_pos.begin(), h_pos.end(), pos.begin());
  }
  //Increase spatial coherence in memory
  pd->sortParticles();
  return pd;
}

// Initializes and returns a Force Coupling Methor Integrator module
auto createIntegrator(std::shared_ptr<ParticleData> pd, Box box){
  using Scheme = BDHI::EulerMaruyama<BDHI::FCM>;
  //using Scheme = BD::EulerMaruyama;
  Scheme::Parameters par;
  par.temperature = 1;
  par.viscosity = 1;
  par.hydrodynamicRadius =  4;
  par.dt = 0.01;
  par.box = box;
  auto integrator = std::make_shared<Scheme>(pd, par);
  return integrator;
}

// Forwards the simulation and prints particles every once in a while
// This function works for any Integrator module
void runSimulation(std::shared_ptr<ParticleData> pd, std::shared_ptr<Integrator> bdhi){
  std::ofstream out("/dev/stdout");
  Timer tim; tim.tic();
  int numberSteps = 2000;
  int printSteps  = 200;
  forj(0, numberSteps){
    if(j%printSteps==0){
      auto pos = pd->getPos(access::cpu, access::read);
      out<<"#"<<std::endl;
      for(auto p: pos)out<<make_real3(p)<<" 0.7 0\n";
    }
    bdhi->forwardTime();
  }
  auto totalTime = tim.toc();
  System::log<System::MESSAGE>("mean FPS: %.2f", numberSteps/totalTime);
}

// Lets create a simulation and run it.
// We need a particle container and an Integrator. We will also add an
// Interactor to the Integrator.
//
// ParticleData
//      ^
//     / \
//    /   \
//   / 	Interactor: GravityAndWall
//  /     /
// /     v addInteractor()
//Integrator: FCM Hydrodynamics
int main(int argc, char *argv[]){
  {
    Box box({256, 128, 160});
    auto pd = createParticles(); //Particle container
    auto bdhi = createIntegrator(pd, box); //Integrator
    auto gravity = std::make_shared<GravityAndWall>(pd, -box.boxSize.z*0.5); //Interactor
    bdhi->addInteractor(gravity);
    //bdhi->addInteractor(createTPPoissonInteractor(pd));
    runSimulation(pd, bdhi);
  }
  return 0;
}


//bdhi->addInteractor(createTPPoissonInteractor(pd));
//#include <Interactor/SpectralEwaldPoisson.cuh>
// auto createTPPoissonInteractor(std::shared_ptr<ParticleData> pd){
//   {
//     auto charges = pd->getCharge(access::cpu, access::write);
//     std::fill(charges.begin(), charges.end(), 1);
//   }
//   Poisson::Parameters par;
//   par.box = Box({256, 128, 160});
//   //Permittivity
//   par.epsilon = 1;
//   //Gaussian width of the sources
//   par.gw = 4.0;
//   //Overall tolerance of the algorithm
//   par.tolerance = 1e-2;
//   //If a splitting parameter is passed
//   // the code will run in Ewald split mode
//   //Otherwise, the non Ewald version will be used
//   //par.split = 1.0;
//   return std::make_shared<Poisson>(pd, par);
// }
//
